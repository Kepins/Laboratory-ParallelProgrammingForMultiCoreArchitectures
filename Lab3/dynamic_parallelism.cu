#include "hip/hip_runtime.h"
/*
CUDA - dynamic parallelism sample
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
		printf("\n%s\n",s); 
		exit(EXIT_FAILURE);   
}


__device__ void swap(int* a, int* b){
	int temp = *a;
	*a = *b;
	*b = temp;
}

__device__ int partition(int* arr, int low, int high) {
    int pivot = arr[high];
    int i = low - 1;
    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[high]);  
    return i + 1;
}


__global__ 
void quickSort(int* d_array, int low, int high) {
	if (low < high) {
        int pivot = partition(d_array, low, high);

        quickSort<<<1,1>>>(d_array, low, pivot - 1);
        quickSort<<<1,1>>>(d_array, pivot + 1, high);
    }
}

void generateRandomNumbers(int *arr, int n) {
	srand(time(NULL));

    for (int i = 0; i < n; i++) {
        arr[i] = -10000 + rand() % (10000 - (-10000) + 1);
	}

}

void printArray(int* arr, int n){
	for(int i=0;i<n;i++){
		printf("%d: %d\n", i, arr[i]);
	}
}

int main(int argc,char **argv) {
	float milliseconds;
	const int N = 100000;
	int* h_array = (int* )malloc(sizeof(int) * N);
	generateRandomNumbers(h_array, N);

	int* d_array;

	printf("-------------INITIAL-------------\n");
	printArray(h_array, N);

	hipEvent_t start, stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	hipMalloc((void **)&d_array, N * sizeof(int));
	hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

	//run kernel on GPU 
	quickSort<<<1, 1>>>(d_array, 0, N-1);

	hipMemcpy(h_array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
    // Wait for the stop event to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    hipEventElapsedTime(&milliseconds, start, stop);

	printf("-------------SORTED-------------\n");
	printArray(h_array, N);

	printf("Kernel execution time: %.3f ms\n", milliseconds);

	free(h_array);
    hipFree(d_array);
}
