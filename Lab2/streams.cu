#include "hip/hip_runtime.h"
/*
CUDA - prepare the histogram of N numbers in range of <a;b> where a and b should be integers
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

__global__ void computeHistogram(int *d_input, int *d_output, int N, int A, int B, int streamChunk, int streamId) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x+streamId*streamChunk;
    if (idx < N  && idx < (streamId + 1) * streamChunk) {
        // Calculate the bin index for the data value
        int resultIdx = (d_input[idx] - A);
        atomicAdd(&d_output[resultIdx], 1); // Atomic update to avoid race conditions
    }
}

void generateRandomNumbers(int *arr, int N, int A, int B) {
	srand(time(NULL));
    for (int i = 0; i < N; i++) {
        arr[i] = A + rand() % (B - A);
    }
}

int main(int argc,char **argv) {
    ///define number of streams
    int numberOfStreams = 4;
    hipEvent_t start, stop;
    float milliseconds = 0;

    //create streams
    hipStream_t streams[numberOfStreams];
    for(int i=0;i<numberOfStreams;i++) {
        if (hipSuccess!=hipStreamCreate(&streams[i])) {
            errorexit("Error creating stream");
        }
    }

    int N,A,B;
    
    printf("Enter number of elements: \n");
    scanf("%d", &N);

	printf("Enter A value (start range): \n");
    scanf("%d", &A);

    printf("Enter B value (end range): \n");
    scanf("%d", &B);

	//int *randomNumbers = (int *)malloc(N * sizeof(int));
    // if (randomNumbers == NULL) {
    //     printf("Memory allocation failed.\n");
    //     return 1;
    // }
    int * randomNumbers;
    hipHostAlloc(&randomNumbers, N * sizeof(int), hipHostMallocDefault);
    

	generateRandomNumbers(randomNumbers, N,A,B);

    //get number of chunks to operate per stream
    int streamChunk = 1 + ((N - 1)/numberOfStreams);

    printf("Stream chunk is %d \n", streamChunk);
    //define kernel size per stream
    int threadsinblock=1024;
    int blocksingrid=1+((streamChunk-1)/threadsinblock); 
    printf("The kernel will run with: %d blocks\n", blocksingrid);

	int *h_result, *d_result, *d_input;

    // (int *)calloc((B-A), sizeof(int));
	hipHostAlloc(&h_result, (B-A) * sizeof(int), hipHostMallocDefault);

	if (h_result == NULL) {
        printf("Memory allocation failed.\n");
        return 1;
    }

	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	hipMalloc((void **)&d_input, N * sizeof(int));
    hipMalloc((void **)&d_result, (B-A) * sizeof(int));

    // Initialize device histogram to 0
    hipMemset(d_result, 0, (B-A) * sizeof(int));

    //computeHistogram<<<blocksingrid, threadsinblock>>>(d_input, d_result, N, A, B);

    for(int i=0; i<numberOfStreams; i++) {
        hipMemcpyAsync(&d_input[streamChunk*i], &randomNumbers[streamChunk*i], streamChunk*sizeof(int), hipMemcpyHostToDevice, streams[i]);   
        computeHistogram<<<blocksingrid, threadsinblock, threadsinblock*sizeof(int), streams[i]>>>(d_input, d_result, N, A, B, streamChunk, i);
    }
    hipDeviceSynchronize();

    // Copy the histogram result back to the host
    hipMemcpy(h_result, d_result, (B-A) * sizeof(int), hipMemcpyDeviceToHost);


    hipEventRecord(stop, 0);

    // Wait for the stop event to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Print the histogram
    printf("Histogram:\n");
    for (int i = 0; i < B-A; i++) {
        printf("%d occures %d\n", i, h_result[i]);
    }

    // Print execution time
	printf("Kernel execution time: %.3f ms\n", milliseconds);

    // Free allocated memory
    hipHostFree(randomNumbers);
    hipHostFree(h_result);
    hipFree(d_input);
    hipFree(d_result);

    return 0;
}