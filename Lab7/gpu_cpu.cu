#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define VECTOR_SIZE 2000000000
#define THREADS_PER_BLOCK 256
#define MAX_BLOCK_SIZE 800000
#define CHUNK_SIZE ((int64_t)THREADS_PER_BLOCK * MAX_BLOCK_SIZE)
#define CPU_CHUNK_SIZE CHUNK_SIZE / 25600000

// Macro for checking CUDA errors
#define CUDA_CHECK(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA error in file '%s' at line %d: %s\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err));                \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }


// Shared offset variable
int64_t offset = 0;

// Function to compute and update the offset
int64_t get_offset(int64_t chunk_size, int64_t max_size) {
    int64_t current_offset;

    #pragma omp critical
    {
        current_offset = offset;
        offset += chunk_size;
    }

    return current_offset;
}

__global__ void collatz_iterations(const int64_t *input, int64_t *output, int64_t size) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        int64_t n = input[idx];
        int64_t steps = 0;

        while (n != 1) {
            if (n % 2 == 0)
                n /= 2;
            else
                n = 3 * n + 1;
            steps++;
        }
        output[idx] = steps;
    }
}

void cpu_collatz_iterations(const int64_t *input, int64_t *output, int64_t size){
    #pragma omp dynamic for
    for(int idx=0;idx<size;idx++){
        int64_t n = input[idx];
        int64_t steps = 0;

        while (n != 1) {
            if (n % 2 == 0)
                n /= 2;
            else
                n = 3 * n + 1;
            steps++;
        }
        output[idx] = steps;
    }
}

int main() {
    // Host memory allocation
    int64_t *h_vector = (int64_t *)malloc(VECTOR_SIZE * sizeof(int64_t));
    int64_t *h_output = (int64_t *)malloc(VECTOR_SIZE * sizeof(int64_t));

    if (!h_vector || !h_output) {
        fprintf(stderr, "Host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // Fill the input vector
    #pragma omp parallel
    for (int64_t i = 0; i < VECTOR_SIZE; i++) {
        h_vector[i] = (80000000 + i);
    }

    // Device memory allocation for one chunk
    int64_t *d_vector1, *d_vector2;
    int64_t *d_output1, *d_output2;
    CUDA_CHECK(hipMalloc((void **)&d_vector1, CHUNK_SIZE * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc((void **)&d_vector2, CHUNK_SIZE * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc((void **)&d_output1, CHUNK_SIZE * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc((void **)&d_output2, CHUNK_SIZE * sizeof(int64_t)));

    // Create CUDA streams
    hipStream_t stream1, stream2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));

    double start = omp_get_wtime();

    
    #pragma omp parallel num_threads(2)
    {
        int my_tid = omp_get_thread_num();
        if (my_tid == 0){
            while(1){
                int64_t offset = get_offset(CHUNK_SIZE, VECTOR_SIZE);
                if (offset >= VECTOR_SIZE){
                    break;
                }

                int64_t current_chunk_size = (VECTOR_SIZE - offset) < CHUNK_SIZE ? (VECTOR_SIZE - offset) : CHUNK_SIZE;

                int64_t blocks = ((current_chunk_size - 1) / THREADS_PER_BLOCK) + 1;

                if (my_tid % 2 == 0) {
                    // Process with stream1
                    CUDA_CHECK(hipStreamSynchronize(stream1));
                    CUDA_CHECK(hipMemcpyAsync(d_vector1, h_vector + offset, current_chunk_size * sizeof(int64_t), hipMemcpyHostToDevice, stream1));
                    collatz_iterations<<<blocks, THREADS_PER_BLOCK, 0, stream1>>>(d_vector1, d_output1, current_chunk_size);
                    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
                    CUDA_CHECK(hipMemcpyAsync(h_output + offset, d_output1, current_chunk_size * sizeof(int64_t), hipMemcpyDeviceToHost, stream1));
                } else {
                    // Process with stream2
                    CUDA_CHECK(hipStreamSynchronize(stream2));
                    CUDA_CHECK(hipMemcpyAsync(d_vector2, h_vector + offset, current_chunk_size * sizeof(int64_t), hipMemcpyHostToDevice, stream2));
                    collatz_iterations<<<blocks, THREADS_PER_BLOCK, 0, stream2>>>(d_vector2, d_output2, current_chunk_size);
                    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
                    CUDA_CHECK(hipMemcpyAsync(h_output + offset, d_output2, current_chunk_size * sizeof(int64_t), hipMemcpyDeviceToHost, stream2));
                }
            }
        }
        else {
            while(1){
                int64_t offset = get_offset(CPU_CHUNK_SIZE, VECTOR_SIZE);
                if (offset >= VECTOR_SIZE){
                    break;
                }
                int64_t current_chunk_size = (VECTOR_SIZE - offset) < CPU_CHUNK_SIZE ? (VECTOR_SIZE - offset) : CPU_CHUNK_SIZE;
                cpu_collatz_iterations(h_vector + offset, h_output + offset, current_chunk_size);
            }
        }
    }
    

    // Synchronize streams to ensure all operations are complete
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));

    double end = omp_get_wtime();
    

    // Free resources
    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));
    CUDA_CHECK(hipFree(d_vector1));
    CUDA_CHECK(hipFree(d_vector2));
    CUDA_CHECK(hipFree(d_output1));
    CUDA_CHECK(hipFree(d_output2));
    free(h_vector);
    free(h_output);

    printf("gpu_cpu.c took %lf seconds\n", end-start);
    return EXIT_SUCCESS;
}
